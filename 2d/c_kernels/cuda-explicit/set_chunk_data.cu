#include "hip/hip_runtime.h"
#include "../../ABFT/GPU/double_vector.cuh"

__global__ void set_chunk_data_vertices( 
        int x, int y, int halo_depth, double dx, double dy, double x_min,
        double y_min, double_vector vertex_x, double_vector vertex_y,
        double_vector vertex_dx, double_vector vertex_dy, uint32_t size_vertex_x, uint32_t size_vertex_y)
{
  	INIT_DV_WRITE(vertex_x);
  	INIT_DV_WRITE(vertex_y);
  	INIT_DV_WRITE(vertex_dx);
  	INIT_DV_WRITE(vertex_dy);
    const int start_gid = WIDE_SIZE_DV * (blockIdx.x*blockDim.x+threadIdx.x);
    for(uint32_t gid = start_gid, offset = 0; offset < WIDE_SIZE_DV; offset++, gid++)
    {
        if(gid < x+1)
        {
            dv_set_value_s_new(vertex_x, x_min + dx*(gid-halo_depth), gid, 0, size_vertex_x);
            dv_set_value_s_new(vertex_dx, dx, gid, 0, size_vertex_x);
        }
    }

    for(uint32_t gid = start_gid, offset = 0; offset < WIDE_SIZE_DV; offset++, gid++)
    {
        if(gid < y+1)
        {
            dv_set_value_s_new(vertex_y, y_min + dy*(gid-halo_depth), 0, gid, size_vertex_y);
            dv_set_value_s_new(vertex_dy, dy, 0, gid, size_vertex_y);
        }
    }
  	DV_FLUSH_WRITES_S_NEW(vertex_x, size_vertex_x);
  	DV_FLUSH_WRITES_S_NEW(vertex_y, size_vertex_y);
  	DV_FLUSH_WRITES_S_NEW(vertex_dx, size_vertex_x);
  	DV_FLUSH_WRITES_S_NEW(vertex_dy, size_vertex_y);
}

// Extended kernel for the chunk initialisation
__global__ void set_chunk_data( 
        int dim_x, int dim_y, double dx, double dy, double_vector cell_x, double_vector cell_y,
      double_vector cell_dx, double_vector cell_dy, double_vector vertex_x, double_vector vertex_y,
    double_vector volume, double_vector x_area, double_vector y_area,
    uint32_t size_vertex_x, uint32_t size_vertex_y, uint32_t size_cell_x, uint32_t size_cell_y,
    uint32_t size_x_area, uint32_t size_y_area, uint32_t size_x)
{
    const int start_gid = WIDE_SIZE_DV * (blockIdx.x*blockDim.x+threadIdx.x);
    SET_SIZE_X(size_x);
    INIT_DV_READ(vertex_x);
    INIT_DV_READ(vertex_y);
    INIT_DV_WRITE(cell_x);
    INIT_DV_WRITE(cell_dx);
    INIT_DV_WRITE(cell_y);
    INIT_DV_WRITE(cell_dy);
    INIT_DV_WRITE(volume);
    INIT_DV_WRITE(x_area);
    INIT_DV_WRITE(y_area);

    for(uint32_t gid = start_gid, offset = 0; offset < WIDE_SIZE_DV; offset++, gid++)
    {
        if(gid < dim_x)
        {
    		    dv_set_value_s_new(cell_x, 0.5*(dv_get_value_s_new(vertex_x, gid, 0, size_vertex_x)
                                        +dv_get_value_s_new(vertex_x, gid+1, 0, size_vertex_x)), gid, 0, size_cell_x);
    		    dv_set_value_s_new(cell_dx, dx, gid, 0, size_cell_x);
        }
    }

    for(uint32_t gid = start_gid, offset = 0; offset < WIDE_SIZE_DV; offset++, gid++)
    {
        if(gid < dim_y)
        {
            dv_set_value_s_new(cell_y, 0.5*(dv_get_value_s_new(vertex_y, 0, gid, size_vertex_y)
                                            +dv_get_value_s_new(vertex_y, 0, gid+1, size_vertex_y)), 0, gid, size_cell_y);
            dv_set_value_s_new(cell_dy, dy, 0, gid, size_cell_y);
        }
    }

    for(uint32_t gid = start_gid, offset = 0; offset < WIDE_SIZE_DV; offset++, gid++)
    {
        if(gid < dim_x*dim_y)
        {
            uint32_t x = gid % dim_x;
            uint32_t y = gid / dim_x;
          	dv_set_value(volume, dx*dy, x, y);
        }
    }

    for(uint32_t gid = start_gid, offset = 0; offset < WIDE_SIZE_DV; offset++, gid++)
    {
        if(gid < (dim_x+1)*dim_y)
        {
            uint32_t x = gid % (dim_x + 1);
            uint32_t y = gid / (dim_x + 1);
    		    dv_set_value_s_new(x_area, dy, x, y, size_x_area);
        }
    }

    for(uint32_t gid = start_gid, offset = 0; offset < WIDE_SIZE_DV; offset++, gid++)
    {
        if(gid < dim_x*(dim_y+1))
        {
            uint32_t x = gid % dim_x;
            uint32_t y = gid / dim_x;
            dv_set_value_s_new(x_area, dy, x, y, size_y_area);
        }
    }

  	DV_FLUSH_WRITES_S(cell_x, size_cell_x);
  	DV_FLUSH_WRITES_S(cell_dx, size_cell_x);
  	DV_FLUSH_WRITES_S(cell_y, size_cell_y);
  	DV_FLUSH_WRITES_S(cell_dy, size_cell_y);
    DV_FLUSH_WRITES(volume);
    DV_FLUSH_WRITES_S(x_area, size_x_area);
    DV_FLUSH_WRITES_S(y_area, size_y_area);
}

