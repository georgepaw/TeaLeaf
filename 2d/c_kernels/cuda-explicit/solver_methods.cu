#include "hip/hip_runtime.h"
#include <stdint.h>
#include "cuknl_shared.h"
#include "../../shared.h"
#include "../../ABFT/GPU/abft_common.cuh"

#include "../../ABFT/GPU/csr_matrix.cuh"

__global__ void sum_reduce(
        const int n, double* buffer);

void sum_reduce_buffer(
        double* buffer, double* result, int len)
{
    while(len > 1)
    {
        int num_blocks = ceil(len / (double)BLOCK_SIZE);
        sum_reduce<<<num_blocks, BLOCK_SIZE>>>(len, buffer);
        len = num_blocks;
    }

    hipMemcpy(result, buffer, sizeof(double), hipMemcpyDeviceToHost);
    check_errors(__LINE__, __FILE__);
}

__global__ void copy_u(
        const int x_inner,
        const int y_inner,
        const int halo_depth,
        const double* src,
        double* dest)
{
    const int gid = threadIdx.x+blockIdx.x*blockDim.x;
    if(gid >= x_inner*y_inner) return;

    const int x = x_inner + 2*halo_depth;
    const int col = gid % x_inner;
    const int row = gid / x_inner; 
    const int off0 = halo_depth*(x + 1);
    const int index = off0 + col + row*x;

    dest[index] = src[index];	
}

__global__ void calculate_residual(
        const int x_inner,
        const int y_inner,
        const int halo_depth,
        const double* u,
        const double* u0,
        uint32_t* row_index,
        uint32_t* col_index,
        double* non_zeros,
        double* r)
{
    INIT_CSR_ELEMENTS();
    const int gid = threadIdx.x+blockIdx.x*blockDim.x;
    if(gid >= x_inner*y_inner) return;

    const int x = x_inner + 2*halo_depth;
    const int col = gid % x_inner;
    const int row = gid / x_inner; 
    const int off0 = halo_depth*(x + 1);
    const int index = off0 + col + row*x;

    const uint32_t row_begin = row_index[index];
    const uint32_t row_end   = row_index[index+1];

    double smvp = 0.0;

    csr_prefetch_csr_elements(col_index, non_zeros, row_begin);
    for (uint32_t idx = row_begin, i = 0; idx < row_end; idx++, i++)
    {
        uint32_t col;
        double val;
        csr_get_csr_element(col_index, non_zeros, &col, &val, idx);
        smvp += val * u[col];
    }
    r[index] = u0[index] - smvp;
}

__global__ void calculate_2norm(
        const int x_inner,
        const int y_inner,
        const int halo_depth,
        const double* src,
        double* norm)
{
    __shared__ double norm_shared[BLOCK_SIZE];
    norm_shared[threadIdx.x] = 0.0;

    const int gid = threadIdx.x+blockIdx.x*blockDim.x;

    if(gid >= x_inner*y_inner) return;

    const int x = x_inner + 2*halo_depth;
    const int col = gid % x_inner;
    const int row = gid / x_inner; 
    const int off0 = halo_depth*(x + 1);
    const int index = off0 + col + row*x;

    norm_shared[threadIdx.x] = src[index]*src[index];

    reduce<double, BLOCK_SIZE/2>::run(norm_shared, norm, SUM);
}

__global__ void finalise(
        const int x_inner,
        const int y_inner,
        const int halo_depth,
        const double* density,
        const double* u,
        double* energy)
{
    const int gid = threadIdx.x+blockIdx.x*blockDim.x;
    if(gid >= x_inner*y_inner) return;

    const int x = x_inner + 2*halo_depth;
    const int col = gid % x_inner;
    const int row = gid / x_inner; 
    const int off0 = halo_depth*(x + 1);
    const int index = off0 + col + row*x;

    energy[index] = u[index]/density[index];
}

__global__ void sum_reduce(
        const int n,
        double* buffer)
{
    __shared__ double buffer_shared[BLOCK_SIZE];

    const int gid = threadIdx.x+blockIdx.x*blockDim.x;
    buffer_shared[threadIdx.x] = (gid < n) ? buffer[gid] : 0.0;

    reduce<double, BLOCK_SIZE/2>::run(buffer_shared, buffer, SUM);
}

__global__ void zero_buffer(
        const int x,
        const int y,
        double* buffer)
{
    const int gid = threadIdx.x+blockIdx.x*blockDim.x;

    if(gid < x*y)
    {
        buffer[gid] = 0.0;
    }
}
