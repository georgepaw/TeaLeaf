#include "hip/hip_runtime.h"
#include <stdint.h>
#include "c_kernels.h"
#include "cuknl_shared.h"
#include "../../ABFT/GPU/csr_matrix.cuh"
#include "../../ABFT/GPU/double_vector.cuh"

__global__ void csr_init_rows(
        const int x,
        const int y,
        const int halo_depth,
        uint32_t* rows)
{
    // Necessarily serialised row index calculation
    const uint32_t num_rows = x * y + 1;
    INIT_CSR_INT_VECTOR_SETUP();
    csr_set_row_value(rows, 0, 0, num_rows);
    uint32_t current_row = 0;
    for(int jj = 0; jj < y; ++jj)
    {
        for(int kk = 0; kk < x; ++kk)
        {
            int index = kk + jj*x;
            // Calculate position dependent row count
            int row_count = 5;
            if (jj <    halo_depth || kk <    halo_depth ||
                jj >= y-halo_depth || kk >= x-halo_depth)
            {
              row_count = 0;
            }
            current_row += row_count;
            csr_set_row_value(rows, current_row, index + 1, num_rows);
        }
    }
    CSR_MATRIX_FLUSH_WRITES_INT_VECTOR(rows, num_rows);
}

__global__ void cg_init_u(
        const int dim_x, const int dim_y,
        const uint32_t size_x, const int coefficient,
        double_vector density, double_vector energy1, double_vector u,
        double_vector p, double_vector r, double_vector w)
{
    SET_SIZE_X(size_x);
    INIT_DV_READ(energy1);
    INIT_DV_READ(density);
    INIT_DV_WRITE(p);
    INIT_DV_WRITE(r);
    INIT_DV_WRITE(u);
    INIT_DV_WRITE(w);
    const uint32_t gid = WIDE_SIZE_DV * (threadIdx.x+blockIdx.x*blockDim.x);

    const uint32_t y = gid / dim_x;
    const uint32_t start_x = gid % dim_x;

    for(uint32_t x = start_x, offset = 0; offset < WIDE_SIZE_DV; offset++, x++)
    {
        if(y < dim_y && x < dim_x)
        {
            dv_set_value(p, 0.0, x, y);
            dv_set_value(r, 0.0, x, y);
            dv_set_value(u,
                         dv_get_value(energy1, x, y)*
                         dv_get_value(density, x, y),
                         x, y);

            dv_set_value(w, (coefficient == CONDUCTIVITY)
                ? dv_get_value(density, x, y) : 1.0/dv_get_value(density, x, y), x, y);
        }
    }
    DV_FLUSH_WRITES(p);
    DV_FLUSH_WRITES(r);
    DV_FLUSH_WRITES(u);
    DV_FLUSH_WRITES(w);
}

__global__ void cg_init_k(
        const int x_inner, const int y_inner,
        const int dim_x, const int dim_y, const uint32_t size_x, const int halo_depth,
        double_vector w, double_vector kx, double_vector ky, double rx, double ry)
{
    SET_SIZE_X(size_x);
    INIT_DV_READ(w);
    INIT_DV_WRITE(kx);
    INIT_DV_WRITE(ky);
    const uint32_t gid = WIDE_SIZE_DV * (threadIdx.x+blockIdx.x*blockDim.x);

    const uint32_t y = gid / dim_x + halo_depth;
    const uint32_t start_x = gid % dim_x;

    for(uint32_t x = start_x, offset = 0; offset < WIDE_SIZE_DV; offset++, x++)
    {
        if(halo_depth <= x && x < dim_x - halo_depth)
        {
        	dv_set_value(kx,
                rx*(dv_get_value(w, x - 1, y)+dv_get_value(w, x, y)) /
                (2.0*dv_get_value(w, x - 1, y)*dv_get_value(w, x, y)), x, y);
        	dv_set_value(ky,
                ry*(dv_get_value(w, x, y - 1)+dv_get_value(w, x, y)) /
                (2.0*dv_get_value(w, x, y - 1)*dv_get_value(w, x, y)), x, y);
        }
    }
    DV_FLUSH_WRITES(kx);
    DV_FLUSH_WRITES(ky);
}

__global__ void cg_init_csr(
        const int dim_x, const int dim_y, const uint32_t size_x, const int halo_depth,
        double_vector kx, double_vector ky, uint32_t* row_index,
        uint32_t* col_index, double* non_zeros)
{
    INIT_CSR_INT_VECTOR();
    SET_SIZE_X(size_x);
    INIT_DV_READ(kx);
    INIT_DV_READ(ky);
    const uint32_t gid = threadIdx.x+blockIdx.x*blockDim.x;
    const uint32_t y = gid / dim_x;
    const uint32_t x = gid % dim_x;
    if(y >= dim_y) return;

    uint32_t coef_index;
    csr_get_row_value(row_index, &coef_index, gid);

    if (y <        halo_depth || x <        halo_depth ||
        y >= dim_y-halo_depth || x >= dim_x-halo_depth) return;
    double vals[5] =
    {
        -dv_get_value(ky, x, y),
        -dv_get_value(kx, x, y),
        (1.0 +
            dv_get_value(kx, x + 1, y) + dv_get_value(kx, x, y) +
            dv_get_value(ky, x, y + 1) + dv_get_value(ky, x, y)),
        -dv_get_value(kx, x + 1, y),
        -dv_get_value(ky, x, y + 1)
    };
    uint32_t cols[5] =
    {
        gid-dim_x,
        gid-1,
        gid,
        gid+1,
        gid+dim_x
    };
    csr_set_csr_element_values(col_index, non_zeros, cols, vals, coef_index, 5);
}

__global__ void cg_init_others(
        const int x_inner, const int y_inner,
        const int dim_x, const int dim_y, const uint32_t size_x, const int halo_depth,
        double_vector u, uint32_t* row_index, uint32_t* col_index,
        double* non_zeros, double_vector p, double_vector r, double_vector w, double_vector mi,
        double* rro)
{
    INIT_CSR_ELEMENTS();
    INIT_CSR_INT_VECTOR();
    SET_SIZE_X(size_x);
    INIT_DV_READ(u);
    INIT_DV_WRITE(w);
    INIT_DV_WRITE(r);
    INIT_DV_WRITE(p);
    __shared__ double rro_shared[BLOCK_SIZE];
    rro_shared[threadIdx.x] = 0.0;
    const uint32_t gid = WIDE_SIZE_DV * (threadIdx.x+blockIdx.x*blockDim.x);

    const uint32_t y = gid / dim_x + halo_depth;
    const uint32_t start_x = gid % dim_x;

    for(uint32_t x = start_x, offset = 0; offset < WIDE_SIZE_DV; offset++, x++)
    {
        if(halo_depth <= x && x < dim_x - halo_depth)
        {
            const uint32_t index = x + y * dim_x;

            double smvp = 0.0;

            uint32_t row_begin;
            csr_get_row_value(row_index, &row_begin, index);
            uint32_t row_end;
            csr_get_row_value(row_index, &row_end, index+1);

            csr_prefetch_csr_elements(col_index, non_zeros, row_begin);
            for (uint32_t idx = row_begin, i = 0; idx < row_end; idx++, i++)
            {
                uint32_t col;
                double val;
                csr_get_csr_element(col_index, non_zeros, &col, &val, idx);
                uint32_t t_x = col % dim_x;
                uint32_t t_y = col / dim_x;
                smvp += val * dv_get_value(u, t_x, t_y);
            }

            dv_set_value(w, smvp, x, y);
            double r_val = dv_get_value(u, x, y) - smvp;
            dv_set_value(r, r_val, x, y);
            dv_set_value(p, r_val, x, y);

            rro_shared[threadIdx.x] += r_val*r_val;
        }
    }
    DV_FLUSH_WRITES(w);
    DV_FLUSH_WRITES(r);
    DV_FLUSH_WRITES(p);

    reduce<double, BLOCK_SIZE/2>::run(rro_shared, rro, SUM);
}

__global__ void cg_calc_w_check(
        const int x_inner, const int y_inner,
        const int dim_x, const int dim_y, const uint32_t size_x, const int halo_depth,
        double_vector p, uint32_t* row_index, uint32_t* col_index,
        double* non_zeros, double_vector w, double* pw)
{
    INIT_CSR_ELEMENTS();
    INIT_CSR_INT_VECTOR();
    SET_SIZE_X(size_x);
    INIT_DV_READ(p);
    INIT_DV_WRITE(w);
    const uint32_t gid = WIDE_SIZE_DV * (threadIdx.x+blockIdx.x*blockDim.x);
    __shared__ double pw_shared[BLOCK_SIZE];
    pw_shared[threadIdx.x] = 0.0;

    const uint32_t y = gid / dim_x + halo_depth;
    const uint32_t start_x = gid % dim_x;

    for(uint32_t x = start_x, offset = 0; offset < WIDE_SIZE_DV; offset++, x++)
    {
        if(halo_depth <= x && x < dim_x - halo_depth)
        {
            const uint32_t index = x + y * dim_x;

            double smvp = 0.0;

            uint32_t row_begin;
            csr_get_row_value(row_index, &row_begin, index);
            uint32_t row_end;
            csr_get_row_value(row_index, &row_end, index+1);

            csr_prefetch_csr_elements(col_index, non_zeros, row_begin);
            for (uint32_t idx = row_begin, i = 0; idx < row_end; idx++, i++)
            {
                uint32_t col;
                double val;
                csr_get_csr_element(col_index, non_zeros, &col, &val, idx);
                uint32_t t_x = col % dim_x;
                uint32_t t_y = col / dim_x;
                smvp += val * dv_get_value(p, t_x, t_y);
            }

            dv_set_value(w, smvp, x, y);
            pw_shared[threadIdx.x] += smvp*dv_get_value(p, x, y);
        }
    }
    DV_FLUSH_WRITES(w);

    reduce<double, BLOCK_SIZE/2>::run(pw_shared, pw, SUM);
}

__global__ void cg_calc_w_no_check(
        const int x_inner, const int y_inner,
        const int dim_x, const int dim_y, const uint32_t size_x, const int halo_depth,
        const uint32_t nnz, double_vector p, uint32_t* row_index,
        uint32_t* col_index, double* non_zeros, double_vector w, double* pw)
{
    SET_SIZE_X(size_x);
    INIT_DV_READ(p);
    INIT_DV_WRITE(w);
    const uint32_t gid = WIDE_SIZE_DV * (threadIdx.x+blockIdx.x*blockDim.x);
    __shared__ double pw_shared[BLOCK_SIZE];
    pw_shared[threadIdx.x] = 0.0;

    const uint32_t y = gid / dim_x + halo_depth;
    const uint32_t start_x = gid % dim_x;

    for(uint32_t x = start_x, offset = 0; offset < WIDE_SIZE_DV; offset++, x++)
    {
        if(halo_depth <= x && x < dim_x - halo_depth)
        {
            const uint32_t index = x + y * dim_x;

            double smvp = 0.0;

            uint32_t row_begin;
            csr_get_row_value_no_check(row_index, &row_begin, index, nnz);
            uint32_t row_end;
            csr_get_row_value_no_check(row_index, &row_end, index+1, nnz);

            for (uint32_t idx = row_begin, i = 0; idx < row_end; idx++, i++)
            {
                uint32_t col;
                double val;
                csr_get_csr_element_no_check(col_index, non_zeros, &col, &val, idx, dim_x * dim_y);
                uint32_t t_x = col % dim_x;
                uint32_t t_y = col / dim_x;
                smvp += val * dv_get_value(p, t_x, t_y);
            }

            dv_set_value(w, smvp, x, y);
            pw_shared[threadIdx.x] += smvp*dv_get_value(p, x, y);
        }
    }
    DV_FLUSH_WRITES(w);

    reduce<double, BLOCK_SIZE/2>::run(pw_shared, pw, SUM);
}

__global__ void cg_calc_ur(
        const int x_inner, const int y_inner,
        const int dim_x, const int dim_y, const uint32_t size_x, const int halo_depth,
        const double alpha, double_vector p, double_vector w,
        double_vector u, double_vector r, double* rrn)
{
    SET_SIZE_X(size_x);
    INIT_DV_READ(p);
    INIT_DV_READ(w);
    INIT_DV_READ(u);
    INIT_DV_READ(r);
    INIT_DV_WRITE(u);
    INIT_DV_WRITE(r);
    const uint32_t gid = WIDE_SIZE_DV * (threadIdx.x+blockIdx.x*blockDim.x);
    __shared__ double rrn_shared[BLOCK_SIZE];
    rrn_shared[threadIdx.x] = 0.0;

    const uint32_t y = gid / dim_x + halo_depth;
    const uint32_t start_x = gid % dim_x;
    dv_fetch_manual(p, start_x, y);
    dv_fetch_manual(w, start_x, y);
    dv_fetch_manual(u, start_x, y);
    dv_fetch_manual(r, start_x, y);
    for(uint32_t x = start_x, offset = 0; offset < WIDE_SIZE_DV; offset++, x++)
    {
        if(halo_depth <= x && x < dim_x - halo_depth)
        {
            dv_set_value_manual(u, dv_get_value_manual(u, x, offset, y) + alpha*dv_get_value_manual(p, x, offset, y), x, offset, y);
            double r_temp = dv_get_value_manual(r, x, offset, y) - alpha*dv_get_value_manual(w, x, offset, y);
            dv_set_value_manual(r, r_temp, x, offset, y);
            rrn_shared[threadIdx.x] += r_temp*r_temp;
        }
    }
    dv_flush_manual(u, start_x, y);
    dv_flush_manual(r, start_x, y);

    reduce<double, BLOCK_SIZE/2>::run(rrn_shared, rrn, SUM);
}

__global__ void cg_calc_p(
        const int x_inner, const int y_inner,
        const int dim_x, const int dim_y, const uint32_t size_x, const int halo_depth,
        const double beta, double_vector r, double_vector p)
{
    SET_SIZE_X(size_x);
    INIT_DV_READ(p);
    INIT_DV_READ(r);
    INIT_DV_WRITE(p);
    const uint32_t gid = WIDE_SIZE_DV * (threadIdx.x+blockIdx.x*blockDim.x);

    const uint32_t y = gid / dim_x + halo_depth;
    const uint32_t start_x = gid % dim_x;
    dv_fetch_manual(p, start_x, y);
    dv_fetch_manual(r, start_x, y);
    for(uint32_t x = start_x, offset = 0; offset < WIDE_SIZE_DV; offset++, x++)
    {
        if(halo_depth <= x && x < dim_x - halo_depth)
        {
            double val = beta*dv_get_value_manual(p, x, offset, y) + dv_get_value_manual(r, x, offset, y);
            dv_set_value_manual(p, val, x, offset, y);
        }
    }
    dv_flush_manual(p, start_x, y);
}

__global__ void matrix_check(
        const int x_inner,
        const int y_inner,
        const int halo_depth,
        uint32_t* row_index,
        uint32_t* col_index,
        double* non_zeros)
{
    INIT_CSR_INT_VECTOR();
    INIT_CSR_ELEMENTS();
    const int gid = threadIdx.x+blockIdx.x*blockDim.x;

    if(gid < x_inner*y_inner)
    {
        const int x = x_inner + 2*halo_depth;
        const int col = gid % x_inner;
        const int row = gid / x_inner;
        const int off0 = halo_depth*(x + 1);
        const int index = off0 + col + row*x;

        uint32_t row_begin;
        csr_get_row_value(row_index, &row_begin, index);
        uint32_t row_end;
        csr_get_row_value(row_index, &row_end, index+1);

        csr_prefetch_csr_elements(col_index, non_zeros, row_begin);
        for (uint32_t idx = row_begin, i = 0; idx < row_end; idx++, i++)
        {
            uint32_t col;
            double val;
            csr_get_csr_element(col_index, non_zeros, &col, &val, idx);
        }
    }
}

